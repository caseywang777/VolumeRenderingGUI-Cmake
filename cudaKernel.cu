// CUDA runtime
#include <hip/hip_runtime.h>



#include <stdio.h>

__global__ void cudaKernel(float* visHist, int* rawHistogramRay, float* devOtf, int K, int B, int D)

{
	//int i = blockDim.x * blockIdx.x + threadIdx.x;
	int id = blockIdx.x;

	float localVisHist[128];

    int base = id * D * B;
	int slcSample = 0;
	for( int d = 0; d < D; d++ ){
		for( int b = 0; b < B; b++ ){
			localVisHist[b] += rawHistogramRay[base + d * D + b];
		}
	}

}

 
extern "C" double runCudaKernel( float* visHist, int K, int D, int B, float* otf, int* rawHistogramRays )

{

		FILE* fp = fopen( "output.txt", "w" );

         float* devVisHist = 0;

         float* devOtf = 0;

         int* devRawHistogramRay = 0;

         hipError_t cudaStatus;

 

         //Choose which GPU to run on, change this on a multi-GPU system.

         cudaStatus= hipSetDevice(0);

         if(cudaStatus != hipSuccess) {

                   fprintf(fp,"hipSetDevice failed!  Do you havea CUDA-capable GPU installed?");

                   goto Error;

         }

 

         //Allocate GPU buffers for three vectors (two input, one output)    .

         cudaStatus= hipMalloc((void**)&devVisHist, B * sizeof(float));

         if(cudaStatus != hipSuccess) {

                   fprintf(fp,"hipMalloc failed!");

                   goto Error;

         }

 

         cudaStatus= hipMalloc((void**)&devOtf, B * sizeof(float));

         if(cudaStatus != hipSuccess) {

                   fprintf(fp,"hipMalloc failed!");

                   goto Error;

         }

 

         cudaStatus= hipMalloc((void**)&devRawHistogramRay, K * B * D * sizeof(int));

         if(cudaStatus != hipSuccess) {

                   fprintf(fp,"hipMalloc failed!");

                   goto Error;

         }

 

         //Copy input vectors from host memory to GPU buffers.

         cudaStatus= hipMemcpy( devRawHistogramRay, rawHistogramRays, K * B * D * sizeof(int), hipMemcpyHostToDevice);

         if(cudaStatus != hipSuccess) {

                   fprintf(fp,"hipMemcpy failed!");

                   goto Error;

         }

 

         cudaStatus= hipMemcpy(devOtf, otf, B * sizeof(float), hipMemcpyHostToDevice);

         if(cudaStatus != hipSuccess) {

                   fprintf(fp,"hipMemcpy failed!");

                   goto Error;

         }

 

         //Launch a kernel on the GPU with one thread for each element.
		 fprintf( fp, "before kernel\n " );fflush( fp );
         cudaKernel<<<K,1>>>(devVisHist, devRawHistogramRay, devOtf, K, B, D);
		 fprintf( fp, "after kernel\n " );fflush( fp );
 

         //Check for any errors launching the kernel

         cudaStatus= hipGetLastError();

         if(cudaStatus != hipSuccess) {

                   fprintf(fp,"addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));

                   goto Error;

         }

 

         //cudaDeviceSynchronize waits for the kernel to finish, and returns

         //any errors encountered during the launch.

         cudaStatus= hipDeviceSynchronize();

         if(cudaStatus != hipSuccess) {

                   fprintf(fp,"hipDeviceSynchronize returned error code %d after launchingaddKernel!\n", cudaStatus);

                   goto Error;

         }

 

         //Copy output vector from GPU buffer to host memory.

         cudaStatus= hipMemcpy(visHist, devVisHist, B * sizeof(float), hipMemcpyDeviceToHost);

         if(cudaStatus != hipSuccess) {

                   fprintf(fp,"hipMemcpy failed!");

                   goto Error;

         }

		 //output result
		 fprintf( fp, "before output\n " );fflush( fp );
		 for( int i=0; i<B; i++ ){
			 fprintf(fp, "%d : %f \n", i, visHist[i]);
		 }
		 fprintf( fp, "after output\n " );fflush( fp );
 

Error:

         hipFree(devRawHistogramRay);

         hipFree(devVisHist);

         hipFree(devOtf);

 


         if(cudaStatus != hipSuccess) {

                   fprintf(stderr,"addWithCuda failed!");

                   return 1;

         }

 

}